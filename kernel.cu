#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <chrono>
#include <ctime>


// Forward declarations
__device__ void register_q(int x, int y, int num_queens);
__global__ void N_Queens_Kernel(int num_queens);


// Global variables
const int Nq = 21; // (2147483648 / 8); // N = 1 / 8 maxint32 = 268, 435, 456 queens


// GPU-local variables
__device__ int board[Nq] = { 0 };   // list of queen positions, where board[x] = y
__device__ short occ_col[Nq];       // column occupancy
__device__ short occ_row[Nq];       // row occupancy
__device__ short occ_adiag[2 * Nq]; // ascending diagonal occupancy
__device__ short occ_ddiag[2 * Nq]; // decending diagonal occupancy
__device__ short collision_flag[1] = { 0 }; // Flag raised if any 2 Queens can attack each other


// GPU functions
__device__ void register_q(int x, int y, int num_queens) // Check for collision and add queen to occupancy lists
{
	if (occ_col[x] != 0 || occ_row[y] != 0 || occ_adiag[(x + y)] != 0 || occ_ddiag[num_queens + (x - y)] != 0) {
		collision_flag[0] = 1;
	}
	occ_col[x] = 1;
	occ_row[y] = 1;
	occ_adiag[x + y] = 1;
	occ_ddiag[num_queens + (x - y)] = 1;
}

// GPU kernel
__global__ void N_Queens_Kernel(int num_queens)
{

	int i = (blockDim.x * blockIdx.x + threadIdx.x) + 1;
	int x, y, x1, y1;

	if (num_queens % 2 == 0 && (num_queens - 2) % 6 != 0) { // Case 1, N is even and (N-2) mod 6 is not 0
		if (i > num_queens / 2) {
			return;
		}
		x = i;
		y = 2 * i;
		x1 = num_queens / 2 + i;
		y1 = 2 * i - 1;

		register_q(x - 1, y - 1, num_queens);
		register_q(x1 - 1, y1 - 1, num_queens);

		board[x - 1] = y - 1;
		board[x1 - 1] = y1 - 1;

	}
	else if (num_queens % 2 == 0 && num_queens % 6 != 0) { // Case 2, N is even and N mod 6 is not 0
		if (i > num_queens / 2) {
			return;
		}
		x = i;
		y = 1 + ((2 * (i - 1) + num_queens / 2 - 1) % num_queens);
		x1 = num_queens + 1 - i;
		y1 = num_queens - ((2 * (i - 1) + num_queens / 2 - 1) % num_queens);

		register_q(x - 1, y - 1, num_queens);
		register_q(x1 - 1, y1 - 1, num_queens);

		board[x - 1] = y - 1;
		board[x1 - 1] = y1 - 1;

	}
	else if ((num_queens - 1) % 2 == 0 && (num_queens - 3) % 6 != 0) { // Case 3
		if (i > (num_queens - 1) / 2) {
			return;
		}
		x = i;
		y = 2 * i;
		x1 = (num_queens - 1) / 2 + i;
		y1 = 2 * i - 1;

		register_q(x - 1, y - 1, num_queens - 1);
		register_q(x1 - 1, y1 - 1, num_queens - 1);

		board[x - 1] = y - 1;
		board[x1 - 1] = y1 - 1;

		if (blockIdx.x == 0 && threadIdx.x == 0) {
			board[num_queens - 1] = num_queens - 1;
		}
	}
	else if ((num_queens - 1) % 2 == 0 && (num_queens - 1) % 6 != 0) { // Case 4
		if (i > (num_queens - 1) / 2) {
			return;
		}
		x = i;
		y = 1 + ((2 * (i - 1) + (num_queens - 1) / 2 - 1) % (num_queens - 1));
		x1 = num_queens - i;
		y1 = (num_queens - 1) - ((2 * (i - 1) + (num_queens - 1) / 2 - 1) % (num_queens - 1));

		register_q(x - 1, y - 1, num_queens - 1);
		register_q(x1 - 1, y1 - 1, num_queens - 1);

		board[x - 1] = y - 1;
		board[x1 - 1] = y1 - 1;

		if (blockIdx.x == 0 && threadIdx.x == 0) {
			board[num_queens - 1] = num_queens - 1;
		}
	}
}


int main()
 {
	auto global_start = std::chrono::system_clock::now(); // Program start time

	int* cflag_ptr = 0;
	int* board_ptr = 0;
	short local_flag = 0;

	// Get pointers to GPU buffers
	hipError_t cudaStatus;
	cudaStatus = hipGetSymbolAddress((void**)&cflag_ptr, collision_flag);
	cudaStatus = hipGetSymbolAddress((void**)&board_ptr, board);

	// Allocate CUDA blocks and threads to dispatch
	int threadsPerBlock = 256;
	int blocksPerGrid = (Nq / 2 + threadsPerBlock - 1) / threadsPerBlock;

	// Initialize
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	if (Nq % 2 == 0 && (Nq - 2) % 6 != 0) {
		std::cout << "Case 1" << std::endl;
	}
	else if (Nq % 2 == 0 && Nq % 6 != 0) {
		std::cout << "Case 2" << std::endl;
	}
	else if ((Nq - 1) % 2 == 0 && (Nq - 3) % 6 != 0) {
		std::cout << "Case 3" << std::endl;
	}
	else if ((Nq - 1) % 2 == 0 && (Nq - 1) % 6 != 0) {
		std::cout << "Case 4" << std::endl;
	}

	auto gpu_start = std::chrono::system_clock::now(); // GPU processing start time

	N_Queens_Kernel << <blocksPerGrid, threadsPerBlock >> > (Nq); // Execute GPU code

	// Check for any errors launching the kernels
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// Wait for all cores to terminate
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy verification flag state to host
	cudaStatus = hipMemcpy(&local_flag, cflag_ptr, sizeof(short), hipMemcpyDeviceToHost);

	// Debug
	std::cout << "N = " << Nq << std::endl;
	if (local_flag == 0) {
		std::cout << "Solution verified" << std::endl;
	}
	auto gpu_end = std::chrono::system_clock::now();
	std::chrono::duration<double> gpu_mseconds = (gpu_end - gpu_start) * 1000;
	std::cout << "GPU time (ms): " << gpu_mseconds.count() << std::endl;


	// Copy output vector from GPU buffer to host memory. Only works for N < 30
	if (Nq < 30) {
		int loc_board[Nq];
		cudaStatus = hipMemcpy(loc_board, board_ptr, Nq * sizeof(int), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		std::cout << "Solution: " << std::endl;

		for (int i = 0; i < Nq; i++) {
			std::cout << loc_board[i] << " ";
		}
		std::cout << std::endl;
	}
	else {
		std::cout << "Solution too large to display" << std::endl;
	}

	// Free up all GPU memory
Error:
	hipFree(board);
	hipFree(collision_flag);
	hipFree(occ_col);
	hipFree(occ_row);
	hipFree(occ_adiag);
	hipFree(occ_ddiag);


	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "queens died :(");
		return 1;
	}

	auto global_end = std::chrono::system_clock::now();
	std::chrono::duration<double> elapsed_seconds = global_end - global_start;
	std::cout << "Total exec time (s): " << elapsed_seconds.count() << std::endl;


	// hipDeviceReset must be called before exiting
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}
