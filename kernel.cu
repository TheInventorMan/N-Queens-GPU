#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <chrono>
#include <ctime>


// Forward declarations
__device__ void register_q(int x, int y, int num_queens);
__device__ void case1(int i, int N);
__device__ void case2(int i, int N);
__global__ void N_Queens_Kernel(int num_queens);
__global__ void clearBuffers(int num_queens);


// Global variables
const int MAX_N = (2147483648 / 8); // N = 1/8 maxint32 = (2147483648 / 8) = 268,435,456 queens


// GPU-local variables
__device__ int board[MAX_N] = { 0 };   // list of queen positions, where board[x] = y
__device__ short occ_col[MAX_N];       // column occupancy
__device__ short occ_row[MAX_N];       // row occupancy
__device__ short occ_adiag[2 * MAX_N]; // ascending diagonal occupancy
__device__ short occ_ddiag[2 * MAX_N]; // decending diagonal occupancy
__device__ short collision_flag[1] = { 0 }; // Flag raised if any 2 Queens can attack each other


// GPU functions
__device__ void register_q(int x, int y, int num_queens) // Check for collision and add queen to occupancy lists
{
	if (occ_col[x] != 0 || occ_row[y] != 0 || occ_adiag[(x + y)] != 0 || occ_ddiag[num_queens + (x - y)] != 0) {
		collision_flag[0] = 1;
	}

	occ_col[x] = 1;
	occ_row[y] = 1;
	occ_adiag[x + y] = 1;
	occ_ddiag[num_queens + (x - y)] = 1;

	return;
}

__device__ void case1(int i, int N) {
	int x, y, x1, y1;
	x = i;
	y = 2 * i;
	x1 = N / 2 + i;
	y1 = 2 * i - 1;

	register_q(x - 1, y - 1, N);
	register_q(x1 - 1, y1 - 1, N);

	board[x - 1] = y - 1;
	board[x1 - 1] = y1 - 1;

	return;
}

__device__ void case2(int i, int N) {
	int x, y, x1, y1;
	x = i;
	y = 1 + ((2 * (i - 1) + N / 2 - 1) % N);
	x1 = N + 1 - i;
	y1 = N - ((2 * (i - 1) + N / 2 - 1) % N);

	register_q(x - 1, y - 1, N);
	register_q(x1 - 1, y1 - 1, N);

	board[x - 1] = y - 1;
	board[x1 - 1] = y1 - 1;

	return;
}

// GPU kernel
__global__ void N_Queens_Kernel(int num_queens)
{

	int i = (blockDim.x * blockIdx.x + threadIdx.x) + 1; // Each thread places 2 queens
	
	if (i > (num_queens - num_queens % 2) / 2) {
		return;
	} 

	if (num_queens % 2 == 0 && (num_queens - 2) % 6 != 0) { // Case 1, N is even and (N-2) mod 6 is not 0
		case1(i, num_queens);
	}
	else if (num_queens % 2 == 0 && num_queens % 6 != 0) { // Case 2, N is even and N mod 6 is not 0
		case2(i, num_queens);
	}
	else if ((num_queens - 1) % 2 == 0 && (num_queens - 3) % 6 != 0) { // Case 3, N is odd, and (N-3) mod 6 is not 0
		case1(i, num_queens - 1);
		if (blockIdx.x == 0 && threadIdx.x == 0) {
			board[num_queens - 1] = num_queens - 1;
		}
	}
	else if ((num_queens - 1) % 2 == 0 && (num_queens - 1) % 6 != 0) { // Case 4, N is odd and (N-1) mod 6 is not 0
		case2(i, num_queens - 1);
		if (blockIdx.x == 0 && threadIdx.x == 0) {
			board[num_queens - 1] = num_queens - 1;
		}
	}

	return;
}

__global__ void clearBuffers(int num_queens) {
	int i = (blockDim.x * blockIdx.x + threadIdx.x); // i < n/2

	board[2*i] = 0;
	board[2*i + 1] = 0;

	occ_col[2*i] = 0;
	occ_col[2 * i + 1] = 0;

	occ_row[2 * i] = 0;
	occ_row[2*i + 1] = 0;

	occ_adiag[2 * i] = 0;
	occ_adiag[2 * i + 1] = 0;
	occ_adiag[2 * i + num_queens] = 0;
	occ_adiag[2 * i + 1 + num_queens] = 0;

	occ_ddiag[2 * i] = 0;
	occ_ddiag[2 * i + 1] = 0;
	occ_ddiag[2 * i + num_queens] = 0;
	occ_ddiag[2 * i + 1 + num_queens] = 0;

	if (blockIdx.x == 0 && threadIdx.x == 0) {
		collision_flag[0] = 0;
	}
	
}

void cls() {
	std::cout << "\033[2J\033[1;1H";
}

int main()
{
	using namespace std;

	auto global_start = chrono::system_clock::now(); // Program start time

	// Store pointers to GPU memory locally
	int* cflag_ptr = 0;
	int* board_ptr = 0;
	short local_flag = 0;

	// Get pointers to GPU buffers
	hipError_t cudaStatus;
	cudaStatus = hipGetSymbolAddress((void**)&cflag_ptr, collision_flag);
	cudaStatus = hipGetSymbolAddress((void**)&board_ptr, board);

	// Initialize GPU
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	while (1) {

		cout << "Interactive GPU-Accelerated N-Queens Solver" << endl;
		cout << "Please select an option: " << endl;
		cout << "1 - Solve for N" << endl;
		cout << "2 - Solve for range of N" << endl;
		cout << "3 - Quit" << endl;

		int resp = 0;
		char _;

		cin >> resp;
		if (resp == 3) {
			break;
		}
		else if (resp == 2) {
			cout << "Not implemented, press any key to continue." << endl;
			cin >> _;
			cls();
			continue;

			int lower, upper;

			cout << "Enter lower bound: ";
			cin >> lower;
			cout << "Enter upper bound: ";
			cin >> upper;


		}
		else if (resp == 1) {

			int Nq = 0;

			cout << "Enter number of queens between 4 and " << MAX_N << ": ";
			cin >> Nq;

			if (Nq < 4 || Nq > MAX_N) {
				cls();
				continue;
			}

			cls();

			global_start = chrono::system_clock::now();

			// Allocate CUDA blocks and threads to dispatch
			int threadsPerBlock = 256;
			int blocksPerGrid = (Nq / 2 + threadsPerBlock) / threadsPerBlock;

			cout << "Launching " << blocksPerGrid << " block with " << threadsPerBlock << " threads each." << endl;
			cout << endl;

			// Display case number depending on value of N
			if (Nq % 2 == 0 && (Nq - 2) % 6 != 0) {
				cout << "Computing... (Case 1)" << endl;
			}
			else if (Nq % 2 == 0 && Nq % 6 != 0) {
				cout << "Computing... (Case 2)" << endl;
			}
			else if ((Nq - 1) % 2 == 0 && (Nq - 3) % 6 != 0) {
				cout << "Computing... (Case 3)" << endl;
			}
			else if ((Nq - 1) % 2 == 0 && (Nq - 1) % 6 != 0) {
				cout << "Computing... (Case 4)" << endl;
			}
			cout << endl;

			auto gpu_start = chrono::system_clock::now(); // GPU processing start time

			N_Queens_Kernel << <blocksPerGrid, threadsPerBlock >> > (Nq); // Execute GPU code

			// Check for any errors launching the kernels
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
				goto Error;
			}

			// Wait for all cores to terminate
			cudaStatus = hipDeviceSynchronize();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching N_Queens_Kernel!\n", cudaStatus);
				goto Error;
			}

			// Copy verification flag state to host
			cudaStatus = hipMemcpy(&local_flag, cflag_ptr, sizeof(short), hipMemcpyDeviceToHost);

			// Verbose debug output
			cout << "N = " << Nq << endl;
			if (local_flag == 0) {
				cout << "Solution verified" << endl;
			}
			cout << endl;
			auto gpu_end = chrono::system_clock::now();
			chrono::duration<double> gpu_mseconds = (gpu_end - gpu_start) * 1000;
			cout << "GPU time (ms): " << gpu_mseconds.count() << endl;

			// Copy output vector from GPU buffer to host memory. Only works for N < 32 (arbitrary)
			if (Nq < 32) {
				int loc_board[32];
				cudaStatus = hipMemcpy(loc_board, board_ptr, Nq * sizeof(int), hipMemcpyDeviceToHost);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipMemcpy failed!");
					goto Error;
				}
				cout << endl;
				cout << "Solution: " << endl;

				for (int i = 0; i < Nq; i++) {
					for (int j = 0; j < Nq; j++) {
						if (j == loc_board[i]) {
							cout << "X" << " ";
						}
						else {
							cout << "-" << " ";
						}
					}
					cout << endl;
				}
				cout << endl;
			}
			else {
				double sol_size = Nq * 16;
				string suffix = " bytes.";
				string prefix = "";

				if (sol_size > 1000000000) {
					sol_size /= 1000000000;
					suffix = " GB.";
				}
				else if (sol_size > 1000000) {
					sol_size /= 1000000;
					suffix = " MB.";
				}
				else if (sol_size > 1000) {
					sol_size /= 1000;
					suffix = " KB.";
				}
				else if (sol_size == 0) {
					sol_size = 2.048;
					prefix = ">";
					suffix = " GB.";
				}
				cout << "Solution too large to display. Solution size: " << prefix << sol_size << suffix << endl;
			}
			cout << endl;

			// Clear board and occupancy grid
			clearBuffers << <blocksPerGrid, threadsPerBlock >> > (Nq);

			// Check for any errors launching the kernels
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
				goto Error;
			}

			// Wait for all cores to terminate
			cudaStatus = hipDeviceSynchronize();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching clearBuffers!\n", cudaStatus);
				goto Error;
			}

			// Display total execution time
			auto global_end = chrono::system_clock::now();
			chrono::duration<double> elapsed_mseconds = 1000*(global_end - global_start);
			cout << "Total exec time (ms): " << elapsed_mseconds.count() << endl;

			cout << "Press any key to continue." << endl;
			cin >> _;
			cls();
		}
	}

	// Free up all GPU memory
Error:
	hipFree(board);
	hipFree(collision_flag);
	hipFree(occ_col);
	hipFree(occ_row);
	hipFree(occ_adiag);
	hipFree(occ_ddiag);

	// Ensure no errors on the status flag
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "queens died :(");
		return 1;
	}

	// hipDeviceReset must be called before exiting
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}
