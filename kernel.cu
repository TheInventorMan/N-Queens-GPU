#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <chrono>
#include <ctime>

// Forward declarations
__device__ void register_q(int x, int y, int num_queens);
__global__ void N_Queens_Kernel(int num_queens);

// Global variables
const int Nq = (2147483648 / 8); // N = 1/8 maxint32 = 268,435,456 queens

// GPU-local variables
__device__ int board[Nq] = { 0 };   // list of queen positions, where board[x] = y
__device__ short occ_col[Nq];       // column occupancy
__device__ short occ_row[Nq];       // row occupancy
__device__ short occ_adiag[2 * Nq]; // ascending diagonal occupancy
__device__ short occ_ddiag[2 * Nq]; // decending diagonal occupancy
__device__ short collision_flag[1] = { 0 }; // Flag raised if any 2 Queens can attack each other

// GPU functions
__device__ void register_q(int x, int y, int num_queens) // Check for collision and add queen to occupancy lists
{
	if (occ_col[x] != 0 || occ_row[y] != 0 || occ_adiag[(x + y)] != 0 || occ_ddiag[num_queens + (x - y)] != 0) {
		collision_flag[0] = 1;
	}
	occ_col[x] = 1;
	occ_row[y] = 1;
	occ_adiag[x + y] = 1;
	occ_ddiag[num_queens + (x - y)] = 1;
}

__global__ void N_Queens_Kernel(int num_queens) // GPU runtime 
{
	int i = (blockDim.x * blockIdx.x + threadIdx.x) + 1;
	int x, y, x1, y1;

	if (num_queens % 2 == 0 && (num_queens - 2) % 6 != 0) { // Case 1, N is even and (N-2) mod 6 is not 0
		x = i - 1;
		y = 2 * i - 1;
		x1 = num_queens / 2 + i - 1;
		y1 = 2 * i - 2;

		register_q(x, y, num_queens);
		register_q(x1, y1, num_queens);

		board[x] = y;
		board[x1] = y1;
	}
	else if (num_queens % 2 == 0 && num_queens % 6 != 0) { // Case 2, N is even and N mod 6 is not 0
		x = i - 1;
		y = (2 * i + num_queens / 2 - 3 % num_queens) % num_queens;
		x1 = num_queens - i;
		y1 = num_queens - (2 * i + num_queens / 2 - 3 % num_queens) - 1;
		if (y1 < 0) {
			y1 += num_queens;
		}

		register_q(x, y, num_queens);
		register_q(x1, y1, num_queens);

		board[x] = y;
		board[x1] = y1;
	}
	else {  // Case 3, all other values of N
		x = i - 1;
		y = 2 * i - 1;
		x1 = (num_queens - 1) / 2 + i - 1;
		y1 = 2 * i - 2;

		register_q(x, y, num_queens - 1);
		register_q(x1, y1, num_queens - 1);

		board[x] = y;
		board[x1] = y1;

		if (collision_flag[0] == 1 || occ_ddiag[0] == 1) {
			x = i - 1;
			y = (2 * i + num_queens / 2 - 3 % num_queens) % num_queens;
			x1 = num_queens - 1 - i;
			y1 = num_queens - 1 - (2 * i + (num_queens - 1) / 2 - 3 % (num_queens - 1)) - 1;
			if (y1 < 0) {
				y1 += num_queens - 1;
			}

			register_q(x, y, num_queens - 1);
			register_q(x1, y1, num_queens - 1);

			board[x] = y;
			board[x1] = y1;
		}
		if (blockIdx.x == 0 && threadIdx.x == 0) {
			board[num_queens - 1] = num_queens - 1;
		}

	}

}

int main()
{
	auto global_start = std::chrono::system_clock::now(); // Program start time

	int* cflag_ptr = 0;
	int* board_ptr = 0;
	short local_flag = 0;
	int loc_board[1];

	// Get pointers to GPU buffers
	hipError_t cudaStatus;
	cudaStatus = hipGetSymbolAddress((void**)&cflag_ptr, collision_flag);
	cudaStatus = hipGetSymbolAddress((void**)&board_ptr, board);

	// Allocate CUDA blocks and threads to dispatch
	int threadsPerBlock = 256;
	int blocksPerGrid = (Nq / 2 + threadsPerBlock - 1) / threadsPerBlock;

	// Initialize
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	auto gpu_start = std::chrono::system_clock::now(); // GPU processing start time

	N_Queens_Kernel <<<blocksPerGrid, threadsPerBlock >>> (Nq); // Execute GPU code

	// Check for any errors launching the kernels
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// Wait for all cores to terminate
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy verification flag state to host
	cudaStatus = hipMemcpy(&local_flag, cflag_ptr, sizeof(short), hipMemcpyDeviceToHost);

	auto gpu_end = std::chrono::system_clock::now();
	std::chrono::duration<double> gpu_mseconds = (gpu_end - gpu_start) * 1000;

	std::cout << "N = " << Nq << std::endl;

	if (local_flag == 0) {
		std::cout << "Solution verified" << std::endl;
	}
	std::cout << "GPU time (ms): " << gpu_mseconds.count() << std::endl;


	// Copy output vector from GPU buffer to host memory. ***Does not work for very big N
	//cudaStatus = hipMemcpy(loc_board, board_ptr, Nq * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Free up all GPU memory
Error:
	hipFree(board);
	hipFree(&collision_flag);
	hipFree(occ_col);
	hipFree(occ_row);
	hipFree(occ_adiag);
	hipFree(occ_ddiag);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "queens died :(");
		return 1;
	}

	auto global_end = std::chrono::system_clock::now();
	std::chrono::duration<double> elapsed_seconds = global_end - global_start;

	std::cout << "Total exec time (s): " << elapsed_seconds.count() << std::endl;


	// hipDeviceReset must be called before exiting
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}


