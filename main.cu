#ifndef __CUDA_RUNTIME_H__
#include "hip/hip_runtime.h"
#endif // !"hip/hip_runtime.h"

#ifndef __DEVICE_LAUNCH_PARAMETERS_H__
#include ""
#endif // !__DEVICE_LAUNCH_PARAMETERS_H__

#include <stdio.h>
#include <string>
#include <iostream>
#include <chrono>
#include <ctime>
#include <vector>

#include "main.cuh"

using namespace std;

// Clear screen helper
void cls() {
	std::cout << "\033[2J\033[1;1H";
}

int main()
{
	hipError_t cudaStatus;

	// Store pointers to GPU memory locally
	int* cflag_ptr = getFlagAddr();
	int* board_ptr = getBoardAddr();
	int MAX_N = getMaxN();
	
	// Program start time
	auto global_start = chrono::system_clock::now(); 

	// Initialize GPU
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Loop until user issues quit command
	while (1) {

		cout << "Interactive GPU-Accelerated N-Queens Solver" << endl;
		cout << "Please select an option: " << endl;
		cout << "1 - Solve for N" << endl;
		cout << "2 - Solve for range of N" << endl;
		cout << "3 - Quit" << endl;

		int resp = 0;
		char _;

		cin >> resp;

		// Quit out of program
		if (resp == 3) { 
			break;
		}

		// Solve for range of N
		else if (resp == 2) { 

			int lower, upper;

			cout << "Enter lower bound (4 <= N <= " << MAX_N << "): ";
			cin >> lower;
			cout << "Enter upper bound (" << lower << " < N <= " << MAX_N << "): ";
			cin >> upper;
			cls();

			if (lower >= upper || lower < 4 || upper > MAX_N) {
				continue;
			}

			global_start = chrono::system_clock::now();

			cudaStatus = rangeSolve(lower, upper, cflag_ptr, board_ptr);

			auto global_end = chrono::system_clock::now();
			chrono::duration<double> elapsed_seconds = (global_end - global_start);
			cout << "Total exec time (s): " << elapsed_seconds.count() << endl;

			cout << endl;
			cout << "Press any key to continue." << endl;
			cin >> _;
			cls();

		}

		// Solve for single value of N
		else if (resp == 1) { 

			int Nq = 0;

			cout << "Enter number of queens (4 <= N <= " << MAX_N << "): ";
			cin >> Nq;
			cls();

			if (Nq < 4 || Nq > MAX_N) {
				continue;
			}

			global_start = chrono::system_clock::now();

			cudaStatus = singleSolve(Nq, cflag_ptr, board_ptr);

			// Display total execution time
			auto global_end = chrono::system_clock::now();
			chrono::duration<double> elapsed_mseconds = 1000*(global_end - global_start);
			cout << "Total exec time (ms): " << elapsed_mseconds.count() << endl;

			cout << "Press any key to continue." << endl;
			cin >> _;
			cls();
		}
	}

	// Free up all GPU memory
Error:
	memPurge();

	// Ensure no errors on the status flag
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "queens died :(");
		return 1;
	}

	// hipDeviceReset must be called before exiting
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}